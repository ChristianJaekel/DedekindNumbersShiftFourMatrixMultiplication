#include "hip/hip_runtime.h"
#include "Device.h"
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform_reduce.h>

static void checkCublasErrors(hipblasStatus_t err) {
    switch (err) {
        case HIPBLAS_STATUS_SUCCESS: {
            break;
        }
        case HIPBLAS_STATUS_NOT_INITIALIZED: {
            std::cout << "HIPBLAS_STATUS_NOT_INITIALIZED" << std::endl;
            break;
        }
        case HIPBLAS_STATUS_ALLOC_FAILED: {
            std::cout << "HIPBLAS_STATUS_ALLOC_FAILED" << std::endl;
            break;
        }
        case HIPBLAS_STATUS_INVALID_VALUE: {
            std::cout << "HIPBLAS_STATUS_INVALID_VALUE" << std::endl;
            break;
        }
        case HIPBLAS_STATUS_ARCH_MISMATCH: {
            std::cout << "HIPBLAS_STATUS_ARCH_MISMATCH" << std::endl;
            break;
        }
        case HIPBLAS_STATUS_MAPPING_ERROR: {
            std::cout << "HIPBLAS_STATUS_MAPPING_ERROR" << std::endl;
            break;
        }
        case HIPBLAS_STATUS_EXECUTION_FAILED: {
            std::cout << "HIPBLAS_STATUS_EXECUTION_FAILED" << std::endl;
            break;
        }
        case HIPBLAS_STATUS_INTERNAL_ERROR:
            std::cout << "HIPBLAS_STATUS_INTERNAL_ERROR" << std::endl;
    }
}

DeviceEnumerator::DeviceEnumerator(const std::vector<uint8_t>& idx) {
    // only one device is used
    mNrOfDevices = 1;

    // std::cout << "Device Memory: " << getAvailableGPUMemory() << std::endl;
    // std::cout << std::endl;

    mAs.resize(mNrOfDevices);
    mBs.resize(mNrOfDevices);
    mCs.resize(mNrOfDevices);

    mIdx.resize(mNrOfDevices);
    mIdxCalc.resize(mNrOfDevices);

    for (unsigned i = 0; i < mNrOfDevices; ++i) {
        hipError_t err{hipSetDevice(i)};
        if (err != hipSuccess) {
            std::cout << hipGetErrorString(err) << std::endl;
        }

        mIdx[i]     = toDevice(idx, i);
        mIdxCalc[i] = IndexCalc(mIdx[i]);
    }

    mDistTop.resize(mNrOfDevices);
    mDistBottom.resize(mNrOfDevices);
    mElements.resize(mNrOfDevices);
    mResults.resize(mNrOfDevices);

    mHandles = std::vector<hipblasHandle_t>(mNrOfDevices);
    for (unsigned i = 0; i < mNrOfDevices; ++i) {
        hipblasCreate(&mHandles[i]);
    }
}

void DeviceEnumerator::setTop(const std::vector<uint64_t>& distTopValues) {
    for (unsigned i = 0; i < mNrOfDevices; ++i) {
        mDistTop[i] = toDevice(distTopValues, i);
    }
}

void DeviceEnumerator::setBottom(const std::vector<uint64_t>& distBottomValues) {
    for (unsigned i = 0; i < mNrOfDevices; ++i) {
        mDistBottom[i] = toDevice(distBottomValues, i);
    }
}

void DeviceEnumerator::setElements(const std::vector<uint16_t>& elements) {
    for (unsigned i = 0; i < mNrOfDevices; ++i) {
        mElements[i] = toDevice(elements, i);
    }
}

void DeviceEnumerator::freeElements() {
    for (unsigned i = 0; i < mNrOfDevices; ++i) {
        hipFree(mElements[i]);
    }
}

void DeviceEnumerator::freeTop() {
    for (unsigned i = 0; i < mNrOfDevices; ++i) {
        hipFree(mDistTop[i]);
    }
}

void DeviceEnumerator::freeBottom() {
    for (unsigned i = 0; i < mNrOfDevices; ++i) {
        hipFree(mDistBottom[i]);
    }
}

DeviceEnumerator::~DeviceEnumerator() {
    for (unsigned i = 0; i < mNrOfDevices; ++i) {
        hipFree(mIdx[i]);
        hipblasDestroy(mHandles[i]);
    }
}

size_t DeviceEnumerator::getAvailableGPUMemory() {
    size_t mf, ma;
    hipMemGetInfo(&mf, &ma);
    return ma;
}

double* DeviceEnumerator::initOnDevice(size_t s, unsigned int deviceId) {
    double* d;
    hipSetDevice(deviceId);
    hipMalloc((void**)&d, s * sizeof(double));

    return d;
}

unsigned int DeviceEnumerator::getNumberOfDevices() {
    int i;
    hipGetDeviceCount(&i);
    return static_cast<unsigned int>(i);
}

void DeviceEnumerator::initMatrices(size_t nrOfElements, size_t matrixBatchSize) {
    const size_t s =
        (((nrOfElements + 1) * nrOfElements) / 2 + numberOfThreads - 1) / numberOfThreads;
    for (unsigned i = 0; i < mNrOfDevices; ++i) {
        hipSetDevice(i);
        mAs[i] = initOnDevice(matrixBatchSize, i);
        mBs[i] = initOnDevice(matrixBatchSize, i);
        mCs[i] = initOnDevice(matrixBatchSize, i);

        uint128T* d;
        hipMalloc((void**)&d, s * sizeof(uint128T));
        mResults[i] = thrust::device_pointer_cast(d);
    }

    hipError_t err{hipGetLastError()};
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << std::endl;
    }
}

void DeviceEnumerator::freeMatrices() {
    for (unsigned i = 0; i < mNrOfDevices; ++i) {
        hipSetDevice(i);
        hipFree(mAs[i]);
        hipFree(mBs[i]);
        hipFree(mCs[i]);
        hipFree(mResults[i].get());
    }
}

void DeviceEnumerator::doMatMulStridedBatched(size_t s, int batchCount, double* C, double* A,
                                              double* B, hipblasHandle_t& handle) {
    const double alpha = 1.0;
    const double beta  = 0.0;

    hipblasStatus_t err =
        hipblasDgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, s, s, s, &alpha, A, s, s * s, B,
                                  s, s * s, &beta, C, s, s * s, batchCount);

    checkCublasErrors(err);
}

// - computes the trace of C^2
// - lineare indicees are mapped to the upper triangular matrix
__global__ void kernelTrace(size_t shift, size_t s, double* C, uint128T* result) {
    __shared__ uint128T tmpVals[numberOfThreads];

    if (threadIdx.x == 0) {
        for (unsigned int i = 0; i < numberOfThreads; ++i)
            tmpVals[i] = 0;
    }
    __syncthreads();

    const uint64_t index = static_cast<uint64_t>(blockIdx.x) * static_cast<uint64_t>(blockDim.x) +
                           static_cast<uint64_t>(threadIdx.x);
    uint64_t indAC = static_cast<uint64_t>((-1 + sqrt(static_cast<double>(8 * index + 1))) / 2);
    uint64_t indBD = index - indAC * (indAC + 1) / 2;

    if (indAC < s && indBD < s) {
        const uint128T symWeight = (static_cast<uint64_t>(indAC != indBD) + 1);

        const uint128T sum1 = C[shift + indAC * s + indBD];
        const uint128T sum2 = C[shift + indBD * s + indAC];

        tmpVals[threadIdx.x] = sum1 * sum2 * symWeight;
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        uint128T sum = 0;
        for (unsigned i = 0; i < numberOfThreads; ++i)
            sum += tmpVals[i];

        result[blockIdx.x] = sum;
    }
}

static uint128T computeTrace(double* C, size_t start, size_t end,
                             const std::vector<std::vector<uint64_t>>& abValues, size_t s,
                             unsigned int deviceId, uint128T* results) {
    uint128T sum{};

    uint64_t numOfBlocks = (((s + 1) * s) / 2 + numberOfThreads - 1) / numberOfThreads;

    dim3 dimGrid(numOfBlocks);
    dim3 dimBlock(numberOfThreads);

    hipSetDevice(deviceId);

    for (size_t batch = 0; batch < (end - start); ++batch) {
        kernelTrace<<<dimGrid, dimBlock>>>(batch * s * s, s, C, results);

        const auto trace = thrust::reduce(thrust::device, results, results + numOfBlocks,
                                          static_cast<uint128T>(0), thrust::plus<uint128T>());

        sum += trace * static_cast<uint128T>(abValues[start + batch][2]);

        hipError_t err{hipGetLastError()};
        if (err != hipSuccess) {
            std::cout << hipGetErrorString(err) << std::endl;
        }
    }

    return sum;
}

// - fills matrices A and B with values
// - lineare indicees are mapped to the upper triangular matrix
// - variable names don't match the ones in the paper
__global__ void fillMatrices(uint16_t a, uint16_t b, uint16_t* elements, uint64_t* distTop,
                             uint64_t* distBottom, IndexCalc idxCalc, size_t s, double* A,
                             double* B, size_t shift) {
    const uint64_t index = static_cast<uint64_t>(blockIdx.x) * static_cast<uint64_t>(blockDim.x) +
                           static_cast<uint64_t>(threadIdx.x);
    const uint64_t ind1 =
        static_cast<uint64_t>((-1 + sqrt(static_cast<double>(8 * index + 1))) / 2);
    const uint64_t ind2 = index - ind1 * (ind1 + 1) / 2;

    if (ind1 < s && ind2 < s) {
        const auto c   = elements[ind1];
        const auto aMc = c & a;
        const auto aJc = c | a;
        const auto bMc = c & b;
        const auto bJc = c | b;

        const auto d     = elements[ind2];
        const auto aMcMd = d & aMc;
        const auto bJcJd = d | bJc;
        const auto bMcMd = d & bMc;
        const auto aJcJd = d | aJc;

        const auto alpha = distBottom[idxCalc.index(aMcMd)] * distTop[idxCalc.index(bJcJd)];
        const auto beta  = distBottom[idxCalc.index(bMcMd)] * distTop[idxCalc.index(aJcJd)];

        A[shift + ind1 * s + ind2] = alpha;
        B[shift + ind1 * s + ind2] = beta;
        A[shift + ind2 * s + ind1] = alpha;
        B[shift + ind2 * s + ind1] = beta;
    }
}

static void fillMatrices(size_t start, size_t end,
                         const std::vector<std::vector<uint64_t>>& abValues, size_t s,
                         unsigned int deviceId, double* A, double* B, uint16_t* elements,
                         uint64_t* distTop, uint64_t* distBottom, IndexCalc idxCalc) {
    uint64_t numOfBlocks = (((s + 1) * s) / 2 + numberOfThreads - 1) / numberOfThreads;

    dim3 dimGrid(numOfBlocks);
    dim3 dimBlock(numberOfThreads);

    for (size_t batch = 0; batch < (end - start); ++batch) {
        hipSetDevice(deviceId);
        fillMatrices<<<dimGrid, dimBlock>>>(static_cast<uint16_t>(abValues[start + batch][0]),
                                            static_cast<uint16_t>(abValues[start + batch][1]),
                                            elements, distTop, distBottom, idxCalc, s, A, B,
                                            static_cast<size_t>(s * s * batch));
    }
};

uint128T DeviceEnumerator::doEnumerationGPU(size_t start, size_t end,
                                            const std::vector<std::vector<uint64_t>>& abValues,
                                            size_t s, unsigned int deviceId) {
    const size_t batchCount = end - start;

    fillMatrices(start, end, abValues, s, deviceId, mAs[deviceId], mBs[deviceId],
                 mElements[deviceId], mDistTop[deviceId], mDistBottom[deviceId],
                 mIdxCalc[deviceId]);

    doMatMulStridedBatched(s, batchCount, mCs[deviceId], mAs[deviceId], mBs[deviceId],
                           mHandles[deviceId]);

    hipError_t err{hipGetLastError()};
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << std::endl;
    }

    return computeTrace(mCs[deviceId], start, end, abValues, s, deviceId, mResults[deviceId].get());
}
